// Working implementation

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define INFILE "/home/hpc_group_04/Drogato/project/Parallelizing_Conway/input/input.txt"
#define OUTMAT "/home/hpc_group_04/Drogato/project/Parallelizing_Conway/output/mat.txt"
#define OUTCNT "/home/hpc_group_04/Drogato/project/Parallelizing_Conway/output/cnt.txt"
#define OUTSTREAK "/home/hpc_group_04/Drogato/project/Parallelizing_Conway/output/streak.txt"
#define STATS "/home/hpc_group_04/Drogato/project/Parallelizing_Conway/output/stats_cuda.csv"



__device__ int tot_neighbours(int idx, int block_dim, int *matrix){
    int sum = 0;

    // Cell coordinates
    int x = idx / block_dim;
    int y = idx % block_dim;

    for (int k = -1; k <= 1; k++)
        for (int i = -1; i <= 1; i++)
            if (x + k >= 0 && y + i >= 0 && x + k < block_dim  && y + i < block_dim && (k!=0 || i!=0)) 
                sum += matrix[block_dim * (x + k) + (y + i)];
    

    return sum;
}



__global__ void game_iterations(int *dev_mat, int *dev_streak, int *dev_counter, int *prev, int dim){   
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= dim || y >= dim)
        return;

    int idx = x * dim + y;

    int sum;
    int curr=prev[idx];

    sum = tot_neighbours(idx, dim, prev);

    if (!curr && sum == 3) {
        curr = 1;
        dev_counter[idx]++;
    }
    else if (curr && (sum >= 4 || sum <= 1)) {
        curr = 0;
    }
    else if (curr) {
        dev_counter[idx]++;
        dev_streak[idx]++;
    }
    dev_mat[idx] = curr;
}

// __global__ void update(int *dev_mat, int *prev, int dim) {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;

//     if (x > dim || y > dim)
//         return;

//     int idx = x * dim + y;

//     prev[idx] = dev_mat[idx];
// }


void printer(int *mat, int *streak, int *counter, int N);



int save_stats(int iterations, int table_size, float time, char * slurm_job_id) {
    FILE *file;

    file = fopen(STATS, "a");

    if (file == NULL) {
        printf("Error opening statistics file!\n");
        return 1;
    }

    fprintf(file, "%s,%d,%d,%.3f\n", slurm_job_id, iterations, table_size, time);

    fclose(file);
    return 0;
}

// Function used for debugging
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char * argv[]) {

    if (argc != 2) {
        printf("Number of arguments passed is %d, but should be 2\n", argc);
        return 1;
    }

    int n;
    char *num_elements = getenv("N");
    sscanf(num_elements, "%d", &n);
    int iter;
    char *num_iter = getenv("ITER");
    sscanf(num_iter, "%d", &iter);

    int *mat;
    FILE *fin = fopen(INFILE, "r");
    // FILE *fout = fopen("../output/original.txt", "w");
    int *counter;
    int *streak;

    // matrix allocation
    mat = (int *) malloc(n * n * sizeof(int));
    counter = (int *) malloc(n * n * sizeof(int));
    streak = (int *) malloc(n * n * sizeof(int));
    
    // Initializing matrix
    int value;
    for (int i=0; i < n; i++) {
        for (int j=0; j < n; j++) {
            
            if (fscanf(fin, "%d", &value) == 1) {
                mat[n * i + j] = value;
            } else {
                printf("Error printing matrix at indexes (%d, %d)\n", i, j);
                return 1;
            }
            
        }
    }
    fclose(fin);


    // statistics array initialization into dev mem
    int *dev_counter;
    int *dev_streak;
    int *prev;
    int *dev_mat;

    gpuErrchk(hipMalloc((void **) &dev_counter, n * n * sizeof(int)));
    gpuErrchk(hipMalloc((void **) &dev_streak, n * n * sizeof(int)));
    gpuErrchk(hipMalloc((void **) &prev, n * n * sizeof(int)));
    gpuErrchk(hipMalloc((void**) &dev_mat, n * n * sizeof(int)));

    gpuErrchk(hipMemset(dev_counter, 0x0, n * n * sizeof(int)));
    gpuErrchk(hipMemset(dev_streak, 0x0, n * n * sizeof(int)));
    gpuErrchk(hipMemset(dev_mat, 0x0, n * n * sizeof(int)));

    // copy input to device mem

    gpuErrchk(hipMemcpy(prev, mat, n * n * sizeof(int), hipMemcpyHostToDevice));

    // TODO: device block distribution
    int m = 32;
    dim3 blockSize(m, m, 1);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y, 1);

    // Setting up timer
    hipEvent_t start, stop;
    float elapsedTime;
    gpuErrchk(hipEventCreate(&start)); // create event objects
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(start, 0));

    // launch kernel on GPU
    for (int i=0; i < iter; i++) {
    

        game_iterations<<<gridSize , blockSize>>>(dev_mat, dev_streak, dev_counter, prev, n);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk(hipDeviceSynchronize());
            
        int *temp_matrix = dev_mat;
        dev_mat = prev;
        prev = temp_matrix;
    }

    // Reading timer
    gpuErrchk(hipEventRecord(stop, 0)); // record end event
    gpuErrchk(hipEventSynchronize(stop)); // wait for all device work to complete
    gpuErrchk(hipEventElapsedTime(&elapsedTime, start, stop)); //time between events
    gpuErrchk(hipEventDestroy(start)); //destroy start event
    gpuErrchk(hipEventDestroy(stop)); 


    printf("Total execution time %f ms\n", elapsedTime);

    // gather results
	gpuErrchk(hipMemcpy(mat, prev, n * n * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(counter, dev_counter, n * n * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(streak, dev_streak, n * n * sizeof(int), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(dev_counter));
    gpuErrchk(hipFree(dev_mat));
    gpuErrchk(hipFree(prev));
    gpuErrchk(hipFree(dev_streak));

    // print or save results
    printer(mat, counter, streak, n);

    free(mat);
    free(counter);
    free(streak);


    if (save_stats(iter, n, elapsedTime, argv[1]) != 0) {
        printf("Error saving stats\n");
    }

    return 0;
}


void printer(int *mat, int *counter, int *streak, int N){
    FILE *f_mat, *f_cnt, *f_streak;

    f_mat = fopen(OUTMAT, "w");
    printf("Printing final state of the board...\n");
    for (int i=0; i < N; i++) {
        for (int j=0; j < N; j++) {
            fprintf(f_mat, "%d ", mat[i*N+j]);
        }
        fprintf(f_mat, "\n");
    }

    f_cnt = fopen(OUTCNT, "w");
    printf("Printing overall count of alive generation for single cell...\n");
    for (int i=0; i < N; i++) {
        for (int j=0; j < N; j++) {
            fprintf(f_cnt, "%d ", counter[i*N+j]);
        }
        fprintf(f_cnt, "\n");
    }

    f_streak = fopen(OUTSTREAK, "w");
    printf("Printing maximum consecutive alive generations...\n");
    for (int i=0; i < N; i++) {
        for (int j=0; j < N; j++) {
            fprintf(f_streak, "%d ", streak[i*N+j]);
        }
        fprintf(f_streak, "\n");
    }

    fclose(f_mat);
    fclose(f_cnt);
    fclose(f_streak);
}

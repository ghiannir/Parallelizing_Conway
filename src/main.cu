#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define INFILE "../input/input.txt"
// #define N 1000
// #define ITER 500


__global__ void game_iterations(int *dev_mat, int *dev_streak, int *dev_counter, int iterations, int dim){   
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x > dim || y > dim)
        return;

    int idx = x * dim + y;

    int sum;
    int prev = 0;
    int curr;

    for(int i=0; i < iterations; i++){
        curr = dev_mat[idx];
        // statistics upgrade
        if(curr && prev)
            dev_streak[idx]++;
        if(curr)
            dev_counter[idx]++;
        prev = curr;

        __syncthreads();

        // board update
        sum = tot_neighbours(idx, dim, dev_mat);

        if(!prev && sum == 3)
            dev_mat[idx] = 1;
        else if (prev && (sum >= 4 || sum == 1)){
            dev_mat[idx] = 0;
        }

	    
    }

}

// TODO: farla piu leggibile
__device__ int tot_neighbours(int idx, int block_dim, int *dev_mat){
    int sum = 0;

    // flags for border cells
    int left=0, right=0, up=0, down=0;
    
    if(idx%block_dim == 0)
        left = 1; 
    else if((idx+1)%block_dim == 0)
        right = 1;
    if(idx-block_dim < 0)
        up = 1;
    else if(idx+block_dim >= n*n)
        down=1;
    // sum all existing nearby blocks vlaues
    if(!up){
        sum += dev_mat[idx-block_dim];
        if(!left)
            sum += dev_mat[idx-block_dim-1];
        if(!up)
            sum += dev_mat[idx-block_dim+1];
    }
    if(!down){
        sum += dev_mat[idx+block_dim];
        if(!left)
            sum += dev_mat[idx+block_dim-1];
        if(!up)
            sum += dev_mat[idx+block_dim+1];
    }
    if(!left)
        sum += dev_mat[idx-1];
    if(!up)
        sum += dev_mat[idx+1];
    return sum;
}


void printer(int *mat, int *streak, int *counter, int N);


int main(void){
    int n;
    char *num_elements = getenv("N");
    sscanf(num_elements, "%d", &n);
    int iter;
    char *num_iter = getenv("ITER");
    sscanf(num_iter, "%d", &iter);

    int *mat;
    FILE *fin = fopen(INFILE, "r");
    int *counter;
    int *streak;

    // matrix allocation
    mat = (int *)malloc(n*n*sizeof(int));
    counter = (int *)malloc(n*n*sizeof(int));
    streak = (int *)malloc(n*n*sizeof(int));
    
    for(int i=0; i < n; i++){
        for (int j=0; j < n; j++){
            // reading of the input file and initialization of the matrix
            if(fgetc(fin) == 'X')
                mat[n * i + j] = 1;
            else
                mat[n * i + j] = 0;
        }
    }

    // statistics array initialization into dev mem
    int *dev_counter;
    int *dev_streak;

    hipMalloc((void **)&dev_counter, n * n * sizeof(int));
    hipMalloc((void **)&dev_streak, n * n * sizeof(int));

    hipMemset(dev_counter, 0x0, n * n * sizeof(int));
    hipMemset(dev_streak, 0x0, n * n * sizeof(int));

    // copy input to device mem
    int *dev_mat;

    hipMalloc((void**)&dev_mat, n * n sizeof(int))
    hipMemcpy(dev_mat, mat, n * n * sizeof(int), hipMemcpyHostToDevice);

    // TODO: device block distribution
    dim3 blockSize(n, n);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    // launch kernel on GPU
    // TODO: time measurement
    game_iterations<<<gridSize , blockSize>>>(dev_mat, dev_streak, dev_counter, iter, n);
    
    // gather results
	hipMemcpy(mat, dev_mat, n * n * sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(counter, dev_counter, n * n * sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(streak, dev_streak, n * n * sizeof(int),hipMemcpyDeviceToHost);

    hipFree(dev_counter);
    hipFree(dev_mat);
    hipFree(dev_streak);

    // print or save results
    printer(mat, counter, streak, n);

    free(mat);
    free(counter);
    free(streak);

    fclose(fin);

    return 0;
}


void printer(int *mat, int *streak, int *counter, int N){
    printf("Final state of the board:\n");
    for(int i=0; i < N; i++){
        for(int j=0; j < N; j++){
            printf("%d ", mat[i*N+j]);
        }
        printf("\n");
    }
    
    printf("Overall count of alive generation for single cell:\n");
    for(int i=0; i < N; i++){
        for(int j=0; j < N; j++){
            printf("%d ", counter[i*N+j]);
        }
        printf("\n");
    }

    printf("Maximum consecutive alive generations:\n");
    for(int i=0; i < N; i++){
        for(int j=0; j < N; j++){
            printf("%d ", streak[i*N+j]);
        }
        printf("\n");
    }
}